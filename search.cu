#include <cstring>
#include <filesystem>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


namespace fs = std::filesystem;

struct file {
  char *path;
};

__global__ void search_target_file(file *files, char *target_file_name) {
  int idx = blockIdx.x;
  if (files[idx].path != nullptr) {
    char *temp;
    temp = files[idx].path;
    printf("%c", temp[0]);
  }
}

int main(int argc, char **argv) {
  std::string search_target = argv[1];
  std::cout << "Your target file name : " << search_target << std::endl;

  fs::path current_path = fs::current_path();
  std::cout << "Target directory : " << current_path << std::endl;
  auto dir_iterator = fs::recursive_directory_iterator{current_path};

  std::vector<file> files;
  int file_ctr = 0;
  for (const auto &dir_entry : dir_iterator) {
    auto current_path = dir_entry.path();
    for (auto it = current_path.begin(); it != current_path.end(); ++it) {
      std::string selected_path = it->string();
      file selected_file;
      selected_file.path = (char *)malloc(strlen(selected_path.data()) + 1);
      strcpy(selected_file.path, selected_path.data());
      files.push_back(selected_file);
      file_ctr++;
    }
  }

  file *files_ptr = files.data();
  size_t files_size = sizeof(*files_ptr) * files.size();
  file *device_files = (file *)malloc(files_size);
  hipMalloc(&device_files, files_size);
  hipMemcpy(device_files, files_ptr, files_size, hipMemcpyHostToDevice);

  char *target = search_target.data();
  size_t target_size = sizeof(target);
  char *device_target = (char *)malloc(target_size);
  hipMalloc(&device_target, target_size);
  hipMemcpy(device_target, target, target_size, hipMemcpyHostToDevice);

  search_target_file<<<1000, 1>>>(device_files, device_target);
  hipDeviceSynchronize();
  hipError_t error = hipPeekAtLastError();
  if (error != hipSuccess) {
    std::cout << "Error in kernel code : " << hipGetErrorName(error) << " -> "
              << hipGetErrorString(error) << std::endl;
  }
  hipFree(device_files);
  hipFree(device_target);
  return 0;
}
