#include <cstring>
#include <filesystem>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


namespace fs = std::filesystem;

/*
 * TODO:
 * 1) Since device kernel cannot handle pointer of pointer (afaik) will try
 *  a) using struct, if not works [ ] -> result : TBD
 *  b) using Cuda Thrust Vector [ ] -> result : TBD
 */

__device__ int validate_path_target(char *path, char *target) { return 0; }

__global__ void search_target_file(char *paths, char *target_file_name) {
  int offset_x = threadIdx.x;
  int offset_y = blockIdx.x;
  int idx = offset_x + offset_y * offset_x;
  if (paths[idx] != '\0')
    printf("Current path element -> %c\n", paths[idx]);
}

int main(int argc, char **argv) {
  std::string search_target = argv[1];
  std::cout << "Your target file name : " << search_target << std::endl;

  fs::path current_path = fs::current_path();
  std::cout << "Target directory : " << current_path << std::endl;
  auto dir_iterator = fs::recursive_directory_iterator{current_path};

  std::string all_paths;
  for (const auto &dir_entry : dir_iterator) {
    auto current_path = dir_entry.path();
    for (auto it = current_path.begin(); it != current_path.end(); ++it) {
      all_paths.append("&").append(it->string());
    }
  }
  char *all_paths_chr_ptr = all_paths.data();
  size_t path_size = sizeof(char) * strlen(all_paths_chr_ptr);
  char *device_paths = (char *)malloc(path_size);
  hipMalloc(&device_paths, path_size);
  hipMemcpy(device_paths, all_paths_chr_ptr, path_size,
             hipMemcpyHostToDevice);

  char *target = search_target.data();
  size_t target_size = sizeof(char) * strlen(target);
  char *device_target = (char *)malloc(target_size);
  hipMalloc(&device_target, target_size);
  hipMemcpy(device_target, target, target_size, hipMemcpyHostToDevice);

  search_target_file<<<1000, 1000>>>(device_paths, device_target);
  hipDeviceSynchronize();

  hipError_t error = hipPeekAtLastError();
  if (error != hipSuccess) {
    std::cout << "Error in kernel code : " << hipGetErrorName(error) << " -> "
              << hipGetErrorString(error) << std::endl;
  }

  hipFree(device_paths);
  hipFree(device_target);
  return 0;
}
