#include <algorithm>
#include <filesystem>
#include <iostream>
#include <iterator>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


namespace fs = std::filesystem;

__global__ void search_target_file(char **paths, char *target_file_name) {}

int main(int argc, char **argv) {
  std::string search_target = argv[1];
  std::cout << "Your target file name : " << search_target << std::endl;

  fs::path current_path = fs::current_path();
  std::cout << "Target directory : " << current_path << std::endl;
  auto dir_iterator = fs::recursive_directory_iterator{current_path};

  // Collect all paths to vector
  std::vector<std::string> available_paths;
  for (const auto &dir_entry : dir_iterator) {
    auto current_path = dir_entry.path();
    for (auto iterator = current_path.begin(); iterator != current_path.end();
         ++iterator) {
      available_paths.push_back(iterator->string());
    }
  }

  std::vector<char *> cpaths;
  cpaths.reserve(available_paths.size() + 1);
  std::transform(available_paths.begin(), available_paths.end(),
                 std::back_inserter(cpaths), [](const std::string &s) {
                   char *pc = new char[s.size() + 1];
                   strcpy(pc, s.c_str());
                   return pc;
                 });
  cpaths.push_back(nullptr);
  char **cpaths_ptr = cpaths.data();
  // Only for debug -> check paths in the vector
  for (int i = 0; cpaths_ptr[i] != nullptr; i++) {
    std::cout << cpaths_ptr[i] << std::endl;
  }

  /*
   * TODO: fix the transformation and device (global) func since in device
   * std::string package was not available. So need to pass pointer of char
   * pointer (char **) instead.
   */
  // Transform the vectors to device ready ptrs
  /*std::string *device_paths =
      (std::string *)malloc(sizeof(std::string) * path_size);
  cudaMalloc(&device_paths, sizeof(std::string) * path_size);
  cudaMemcpy(device_paths, paths, sizeof(std::string) * path_size,
             cudaMemcpyHostToDevice);
             */

  // Execute the device function to find the file, pass back to host for bool
  // result (?)
  return 0;
}
