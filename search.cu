#include <cstring>
#include <filesystem>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


/*
 * There's still an issue when assign / copy values from host to device for char
 * **
 * TODO: : figure out the workaround.
 */

namespace fs = std::filesystem;

__device__ int validate_file(char *file, char *target) {
  printf("%c\n", file[0]);
  return 0;
}

__global__ void search_target_file(char **files, char *target_file_name,
                                   int limit) {
  int idx = blockIdx.x;
  char *path = files[idx];
  if (idx < limit) {
    validate_file(path, target_file_name);
  }
}

int main(int argc, char **argv) {
  std::string search_target = argv[1];
  std::cout << "Your target file name : " << search_target << std::endl;

  fs::path current_path = fs::current_path();
  std::cout << "Target directory : " << current_path << std::endl;
  auto dir_iterator = fs::recursive_directory_iterator{current_path};

  std::vector<char *> files;
  int files_char_ctr = 0;
  for (const auto &dir_entry : dir_iterator) {
    auto current_path = dir_entry.path();
    for (auto it = current_path.begin(); it != current_path.end(); ++it) {
      std::string selected_path = it->string();
      files.push_back(selected_path.data());
      files_char_ctr += selected_path.size();
    }
  }

  // Create and assign device pointer for target search file
  char *target = search_target.data();
  size_t target_size = sizeof(target);
  char *device_target = (char *)malloc(target_size);
  hipMalloc(&device_target, target_size);
  hipMemcpy(device_target, target, target_size, hipMemcpyHostToDevice);

  // Create and assign device pointer for path files
  char **device_files_ptr =
      (char **)malloc(sizeof(char) * files.size() * files_char_ctr);
  hipMalloc(&device_files_ptr, sizeof(char) * files.size() * files_char_ctr);
  hipMemcpy(device_files_ptr, files.data(),
             sizeof(char) * files.size() * files_char_ctr,
             hipMemcpyHostToDevice);
  for (int i = 0; i < files.size(); i++) {
    size_t size = sizeof(char) * strlen(files[i]);
    // cudaMalloc(&device_files_ptr[i], size);
    // cudaMemcpy(device_files_ptr[i], files[i], size, cudaMemcpyHostToDevice);
  }

  // Execute device kernel to search the file
  search_target_file<<<1000, 1>>>(device_files_ptr, device_target,
                                  files.size());
  hipDeviceSynchronize();
  hipError_t error = hipPeekAtLastError();
  if (error != hipSuccess) {
    std::cout << "Error in kernel code : " << hipGetErrorName(error) << " -> "
              << hipGetErrorString(error) << std::endl;
  }
  hipFree(device_target);
  return 0;
}
