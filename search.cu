#include <algorithm>
#include <cstring>
#include <filesystem>
#include <iostream>
#include <iterator>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


namespace fs = std::filesystem;

__device__ void validate_path_target(char *path, char *target, bool *result) {
  // TODO: to be improved
  int limit = 100;
  for (int i = 0; i < limit; i++) {
    if (path[i] != target[i]) {
      *result = false;
    }
  }
}

// TODO: fix the implementation, still getting invalid result (might be an error
// @_@)
__global__ void search_target_file(char **paths, char *target_file_name) {
  int idx = blockIdx.x;
  char *current_path = paths[idx];
  /*
   * Notes: since this is device function, some package / library will be not
   * recognize
   * TODO: to be improved
   */
  bool found = true;
  validate_path_target(current_path, target_file_name, &found);
  if (found) {
    printf("Found.\n");
  }
}

int main(int argc, char **argv) {
  std::string search_target = argv[1];
  std::cout << "Your target file name : " << search_target << std::endl;

  fs::path current_path = fs::current_path();
  std::cout << "Target directory : " << current_path << std::endl;
  auto dir_iterator = fs::recursive_directory_iterator{current_path};

  // Collect all paths to vector
  std::vector<std::string> available_paths;
  for (const auto &dir_entry : dir_iterator) {
    auto current_path = dir_entry.path();
    for (auto iterator = current_path.begin(); iterator != current_path.end();
         ++iterator) {
      available_paths.push_back(iterator->string());
    }
  }

  std::vector<char *> cpaths;
  cpaths.reserve(available_paths.size() + 1);
  std::transform(available_paths.begin(), available_paths.end(),
                 std::back_inserter(cpaths), [](const std::string &s) {
                   char *pc = new char[s.size() + 1];
                   strcpy(pc, s.c_str());
                   return pc;
                 });
  cpaths.push_back(nullptr);
  char **cpaths_ptr = cpaths.data();

  // Only for debug -> check paths in the vector
  /*
     for (int i = 0; cpaths_ptr[i] != nullptr; i++) {
    std::cout << cpaths_ptr[i] << std::endl;
  }
  */

  // TODO: found a dynamic way to calculate the right size of cpaths_ptr
  int path_size = 1000;
  char **device_paths = (char **)malloc(path_size);
  hipMalloc(&device_paths, path_size);
  hipMemcpy(device_paths, cpaths_ptr, path_size, hipMemcpyHostToDevice);
  char *target = search_target.data();
  search_target_file<<<1000, 1>>>(device_paths, target);
  hipDeviceSynchronize();

  return 0;
}
